
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<iostream>
#include<fstream>
#include<vector>
#include<string>

#define TILE_WIDTH 16  
#define MAXPOOL_INPUT_FILENAME "input.txt"
#define A_FILENAME "a.txt"
#define B_FILENAME "b.txt"
#define C_FILENAME "c.txt"

using namespace std;

__global__ void maxpool(float *input, float *output, const int input_size, const int filter_size) {
    // input : input_matrix address
    // output : output buffer address
    // input_size : width, height of input matrix
    // filter_size : filter_size of maxpolling
    // all input, output matrices are vectorized

    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    int output_size = input_size / filter_size;

    // out of bound
    if (col >= output_size || row >= output_size)
    {
        return;
    }
    
    // 2D to 1D : (row, col) -> (row * N) + col
    float max_val = input[((row * filter_size) * input_size) + (col * filter_size)];

    for (int i = row * filter_size; i < row * filter_size + filter_size; i++)
    {
        for (int j = col * filter_size; j < col * filter_size + filter_size; j++)
        {
            // update max_val
            max_val = fmaxf(max_val, input[(i * input_size) + j]);
        }
    }
    // assign max value
    output[(row * output_size) + col] = max_val;
}

__global__ void gemm(float *a, float *b, float *c, const float alpha, const float beta, float *output, const int input_size){
    // a, b, c : input matrix address
    // alpha, beta : input constant
    // output : output buffer address
    // input_size : width, height of input matrix
    // all input, output matrices are vectorized

    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x,  by = blockIdx.y;
    int i = blockIdx.x * TILE_WIDTH, j = blockIdx.y * TILE_WIDTH;//add

    int row = by*blockDim.y + ty;
    int col = bx*blockDim.x + tx;
    
    if(row>=input_size ||col>=input_size){
        return;
    }
    
    // allocate 2D tiles in __shared__ memory
    __shared__ float s_a[TILE_WIDTH][TILE_WIDTH];
    __shared__ float s_b[TILE_WIDTH][TILE_WIDTH];
    __shared__ float s_c[TILE_WIDTH][TILE_WIDTH];

    float resultValue = 0.0f;

    // make sure you handle the case when the matrix sizes are not
    // multiple of TILE_WIDTH!
    // loop over the tiles of the input in phases
    for(int p = 0; p < ceilf(input_size/TILE_WIDTH)+1; p++)
    {
        // CHANGE //////////////////////////////////////////////////
        s_a[ty][tx] = 0.0f; // to ignore uneffected values

        // boundary check
        if (row < input_size && (TILE_WIDTH * p + tx) < input_size)
        {
            s_a[ty][tx] = a[row * input_size + TILE_WIDTH * p + tx];
        }

        s_b[ty][tx] = 0.0f; // to ignore uneffected values

        // boundary check
        if (col < input_size && (p * TILE_WIDTH + ty) < input_size)
        {
            s_b[ty][tx] = b[(p * TILE_WIDTH + ty) * input_size + col];
        }
        __syncthreads(); // barrier

        for (int j = 0; j<TILE_WIDTH; j++)
        {
            resultValue += s_a[ty][j] * s_b[j][tx]; // get tile sum for block
        }
        __syncthreads(); // barrier
        // You need to use __syncthreads() a few times
        // to synchronize the threads in a thread block.
    }

    // write out the result to output[row*input_size + col] 
    // CHANGE //////////////////////////////////////////////////
    if (row < input_size && col < input_size)
    {
        int index = (i + tx) + (j + ty)*input_size;
        s_c[ty][tx] = c[index];
        output[index] = alpha * resultValue + beta * s_c[ty][tx];
    }
}


int main(int argc, char **argv)
{
    if(argc < 4) {//check
        cout << "usage : " << argv[0] << " input_size filter_size alpha beta\n" << "example : " << argv[0] << " 100 2 0.5 0.8\n";
        return 1;
    }
    const int input_size = stoi(argv[1]);
    const int filter_size = stoi(argv[2]); // used for maxpooling//check
    const float alpha = stof(argv[3]);
    const float beta = stof(argv[4]);
    const int maxpool_output_size = input_size/filter_size;//check

    // check input_size is power of 2 //16?
    if(input_size == 0 && (input_size & (input_size-1)))
    {
        cout << "input_size must be power of 2\n";//16??
        return 1;
    }

    if(filter_size == 0)
    {
        cout << "filter_size cannot be 0\n";
        return 1;
    }

    //initialize host arrays (array defined in C++)
    float h_maxpool_input[input_size*input_size]; // pointer?
    float h_a[input_size*input_size];
    float h_b[input_size*input_size];
    float h_c[input_size*input_size];
    
    // read input matrices 
    ifstream input_in(MAXPOOL_INPUT_FILENAME); //check ifstream
    ifstream a_in(A_FILENAME);
    ifstream b_in(B_FILENAME);
    ifstream c_in(C_FILENAME);

    //transfer the stream input to host arrays
    for (int i = 0; i < input_size*input_size; ++i)
    {
        input_in >> h_maxpool_input[i];
        a_in >> h_a[i];
        b_in >> h_b[i];
        c_in >> h_c[i];
    }
       
    // set thread, block dimensions
    const dim3 block_size(TILE_WIDTH, TILE_WIDTH);
    const dim3 num_of_maxpool_blocks(maxpool_output_size/block_size.x+1, maxpool_output_size/block_size.y+1);
    const dim3 num_of_blocks(input_size/block_size.x+1, input_size/block_size.y+1);

    // memory allocation for the device arrays (array used in GPU)
    float *d_a, *d_b, *d_c, *d_input, *d_gemm_output, *d_maxpool_output;
    hipMalloc(&d_a, sizeof(float) * input_size * input_size);
    hipMalloc(&d_b, sizeof(float) * input_size * input_size);
    hipMalloc(&d_c, sizeof(float) * input_size * input_size);
    hipMalloc(&d_gemm_output, sizeof(float) * input_size * input_size);
    hipMalloc(&d_input, sizeof(float) * input_size * input_size);
    hipMalloc(&d_maxpool_output, sizeof(float) * maxpool_output_size * maxpool_output_size);
    
    // copy host arrays to device array (so can be used in GPU CUDA kernel)
    hipMemcpy(d_a, h_a, sizeof(float) * input_size * input_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(float) * input_size * input_size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, sizeof(float) * input_size * input_size, hipMemcpyHostToDevice);
    hipMemcpy(d_input, h_maxpool_input, sizeof(float) * input_size * input_size, hipMemcpyHostToDevice);//check

    // launch CUDA kernels

    // First launch gemm kernel using GPU arrays
    gemm<<<num_of_blocks, block_size>>>(d_a, d_b, d_c, alpha, beta, d_gemm_output, input_size);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();//check
    if(error!=hipSuccess)
    {
        fprintf(stderr, "ERROR %s\n", hipGetErrorString(error));
        return 1;
    }
 
    // Then run maxpooling //check
    maxpool<<<num_of_maxpool_blocks, block_size>>>(d_input, d_maxpool_output, input_size, filter_size);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error!=hipSuccess)
    {
        fprintf(stderr, "ERROR %s\n", hipGetErrorString(error));
        return 1;
    }
 
    // allocate output array in host (so host can catch the results from GPU kernel)
    float *h_gemm_output = (float*) malloc (sizeof(float)*input_size*input_size);
    float *h_maxpool_output = (float*) malloc (sizeof(float)*maxpool_output_size*maxpool_output_size);
    
    // copy results from device to host (pass the result from GPU to host)
    hipMemcpy(h_gemm_output, d_gemm_output, sizeof(float)*input_size*input_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_maxpool_output, d_maxpool_output, sizeof(float)*maxpool_output_size*maxpool_output_size, hipMemcpyDeviceToHost);
    
    // prints the results
    cout<<"\n========== GEMM OUTPUT ==========\n";
    for (int i = 0; i < input_size * input_size; i++)
    {
        if(i%input_size==0) cout<<"\n";
        cout<<h_gemm_output[i]<<" ";
    }
    cout<<"\n========== MAXPOOL OUTPUT ==========\n";
    for (int i = 0; i < maxpool_output_size * maxpool_output_size; i++)
    {
        if(i%maxpool_output_size==0) cout<<"\n";
        cout<<h_maxpool_output[i]<<" ";
    }
    cout<<'\n';

    //free everything
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_gemm_output);
    hipFree(d_input);
    hipFree(d_maxpool_output);
    free(h_gemm_output);
    free(h_maxpool_output);

    return 0;
}